#include "hip/hip_runtime.h"
// Copyright 2022 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "lidar_centerpoint/postprocess/circle_nms_kernel.hpp"

#include <lidar_centerpoint/postprocess/postprocess_kernel.hpp>

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

namespace
{
const std::size_t THREADS_PER_BLOCK = 32;
}  // namespace

namespace centerpoint
{

struct is_score_greater
{
  is_score_greater(float t) : t_(t) {}

  __device__ bool operator()(const Box3D & b) { return b.score > t_; }

private:
  float t_{0.0};
};

struct is_kept
{
  __device__ bool operator()(const bool keep) { return keep; }
};

struct score_greater
{
  __device__ bool operator()(const Box3D & lb, const Box3D & rb) { return lb.score > rb.score; }
};

__device__ inline float sigmoid(float x) { return 1.0f / (1.0f + expf(-x)); }

__global__ void generateBoxes3D_kernel(
  const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
  const float * out_rot, const float * out_vel, const float voxel_size_x, const float voxel_size_y,
  const float range_min_x, const float range_min_y, const std::size_t down_grid_size_x,
  const std::size_t down_grid_size_y, const std::size_t downsample_factor, const int class_size,
  const float * yaw_norm_thresholds, Box3D * det_boxes3d)
{
  // generate boxes3d from the outputs of the network.
  // shape of out_*: (N, DOWN_GRID_SIZE_Y, DOWN_GRID_SIZE_X)
  // heatmap: N = class_size, offset: N = 2, z: N = 1, dim: N = 3, rot: N = 2, vel: N = 2
  const auto yi = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  const auto xi = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
  const auto idx = down_grid_size_x * yi + xi;
  const auto down_grid_size = down_grid_size_y * down_grid_size_x;

  if (yi >= down_grid_size_y || xi >= down_grid_size_x) {
    return;
  }

  int label = -1;
  float max_score = -1;
  for (int ci = 0; ci < class_size; ci++) {
    float score = sigmoid(out_heatmap[down_grid_size * ci + idx]);
    if (score > max_score) {
      label = ci;
      max_score = score;
    }
  }

  const float offset_x = out_offset[down_grid_size * 0 + idx];
  const float offset_y = out_offset[down_grid_size * 1 + idx];
  const float x = voxel_size_x * downsample_factor * (xi + offset_x) + range_min_x;
  const float y = voxel_size_y * downsample_factor * (yi + offset_y) + range_min_y;
  const float z = out_z[idx];
  const float w = out_dim[down_grid_size * 0 + idx];
  const float l = out_dim[down_grid_size * 1 + idx];
  const float h = out_dim[down_grid_size * 2 + idx];
  const float yaw_sin = out_rot[down_grid_size * 0 + idx];
  const float yaw_cos = out_rot[down_grid_size * 1 + idx];
  const float yaw_norm = sqrtf(yaw_sin * yaw_sin + yaw_cos * yaw_cos);
  const float vel_x = out_vel[down_grid_size * 0 + idx];
  const float vel_y = out_vel[down_grid_size * 1 + idx];

  det_boxes3d[idx].label = label;
  det_boxes3d[idx].score = yaw_norm >= yaw_norm_thresholds[label] ? max_score : 0.f;
  det_boxes3d[idx].x = x;
  det_boxes3d[idx].y = y;
  det_boxes3d[idx].z = z;
  det_boxes3d[idx].length = expf(l);
  det_boxes3d[idx].width = expf(w);
  det_boxes3d[idx].height = expf(h);
  det_boxes3d[idx].yaw = atan2f(yaw_sin, yaw_cos);
  det_boxes3d[idx].vel_x = vel_x;
  det_boxes3d[idx].vel_y = vel_y;
}

PostProcessCUDA::PostProcessCUDA(const CenterPointConfig & config) : config_(config)
{
  const auto num_raw_boxes3d = config.down_grid_size_y_ * config.down_grid_size_x_;
  boxes3d_d_ = thrust::device_vector<Box3D>(num_raw_boxes3d);
  yaw_norm_thresholds_d_ = thrust::device_vector<float>(
    config_.yaw_norm_thresholds_.begin(), config_.yaw_norm_thresholds_.end());
}

hipError_t PostProcessCUDA::generateDetectedBoxes3D_launch(
  const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
  const float * out_rot, const float * out_vel, std::vector<Box3D> & det_boxes3d,
  hipStream_t stream)
{
  dim3 blocks(
    divup(config_.down_grid_size_y_, THREADS_PER_BLOCK),
    divup(config_.down_grid_size_x_, THREADS_PER_BLOCK));
  dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
  generateBoxes3D_kernel<<<blocks, threads, 0, stream>>>(
    out_heatmap, out_offset, out_z, out_dim, out_rot, out_vel, config_.voxel_size_x_,
    config_.voxel_size_y_, config_.range_min_x_, config_.range_min_y_, config_.down_grid_size_x_,
    config_.down_grid_size_y_, config_.downsample_factor_, config_.class_size_,
    thrust::raw_pointer_cast(yaw_norm_thresholds_d_.data()),
    thrust::raw_pointer_cast(boxes3d_d_.data()));

  // suppress by socre
  const auto num_det_boxes3d = thrust::count_if(
    thrust::device, boxes3d_d_.begin(), boxes3d_d_.end(),
    is_score_greater(config_.score_threshold_));
  if (num_det_boxes3d == 0) {
    return hipGetLastError();
  }
  thrust::device_vector<Box3D> det_boxes3d_d(num_det_boxes3d);
  thrust::copy_if(
    thrust::device, boxes3d_d_.begin(), boxes3d_d_.end(), det_boxes3d_d.begin(),
    is_score_greater(config_.score_threshold_));

  // sort by score
  thrust::sort(det_boxes3d_d.begin(), det_boxes3d_d.end(), score_greater());

  // supress by NMS
  thrust::device_vector<bool> final_keep_mask_d(num_det_boxes3d);
  const auto num_final_det_boxes3d =
    circleNMS(det_boxes3d_d, config_.circle_nms_dist_threshold_, final_keep_mask_d, stream);

  thrust::device_vector<Box3D> final_det_boxes3d_d(num_final_det_boxes3d);
  thrust::copy_if(
    thrust::device, det_boxes3d_d.begin(), det_boxes3d_d.end(), final_keep_mask_d.begin(),
    final_det_boxes3d_d.begin(), is_kept());

  // memcpy device to host
  det_boxes3d.resize(num_final_det_boxes3d);
  thrust::copy(final_det_boxes3d_d.begin(), final_det_boxes3d_d.end(), det_boxes3d.begin());

  return hipGetLastError();
}

}  // namespace centerpoint
