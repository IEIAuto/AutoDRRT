#include "hip/hip_runtime.h"
// Copyright 2022 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "lidar_centerpoint/preprocess/preprocess_kernel.hpp"

#include <lidar_centerpoint/utils.hpp>

namespace
{
const std::size_t MAX_POINT_IN_VOXEL_SIZE = 32;  // the same as max_point_in_voxel_size_ in config
const std::size_t WARPS_PER_BLOCK = 4;
const std::size_t ENCODER_IN_FEATURE_SIZE = 9;  // the same as encoder_in_feature_size_ in config
}  // namespace

namespace centerpoint
{
__global__ void generateVoxels_random_kernel(
  const float * points, size_t points_size, float min_x_range, float max_x_range, float min_y_range,
  float max_y_range, float min_z_range, float max_z_range, float pillar_x_size, float pillar_y_size,
  float pillar_z_size, int grid_y_size, int grid_x_size, unsigned int * mask, float * voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  float4 point = ((float4 *)points)[point_idx];

  if (
    point.x < min_x_range || point.x >= max_x_range || point.y < min_y_range ||
    point.y >= max_y_range || point.z < min_z_range || point.z >= max_z_range)
    return;

  int voxel_idx = floorf((point.x - min_x_range) / pillar_x_size);
  int voxel_idy = floorf((point.y - min_y_range) / pillar_y_size);
  unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if (point_id >= MAX_POINT_IN_VOXEL_SIZE) return;
  float * address = voxels + (voxel_index * MAX_POINT_IN_VOXEL_SIZE + point_id) * 4;
  atomicExch(address + 0, point.x);
  atomicExch(address + 1, point.y);
  atomicExch(address + 2, point.z);
  atomicExch(address + 3, point.w);
}

hipError_t generateVoxels_random_launch(
  const float * points, size_t points_size, float min_x_range, float max_x_range, float min_y_range,
  float max_y_range, float min_z_range, float max_z_range, float pillar_x_size, float pillar_y_size,
  float pillar_z_size, int grid_y_size, int grid_x_size, unsigned int * mask, float * voxels,
  hipStream_t stream)
{
  dim3 blocks((points_size + 256 - 1) / 256);
  dim3 threads(256);
  generateVoxels_random_kernel<<<blocks, threads, 0, stream>>>(
    points, points_size, min_x_range, max_x_range, min_y_range, max_y_range, min_z_range,
    max_z_range, pillar_x_size, pillar_y_size, pillar_z_size, grid_y_size, grid_x_size, mask,
    voxels);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateBaseFeatures_kernel(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, int max_voxel_size,
  unsigned int * pillar_num, float * voxel_features, float * voxel_num, int * voxel_idxs)
{
  unsigned int voxel_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int voxel_idy = blockIdx.y * blockDim.y + threadIdx.y;

  if (voxel_idx >= grid_x_size || voxel_idy >= grid_y_size) return;

  unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;
  unsigned int count = mask[voxel_index];
  if (!(count > 0)) return;
  count = count < MAX_POINT_IN_VOXEL_SIZE ? count : MAX_POINT_IN_VOXEL_SIZE;

  unsigned int current_pillarId = 0;
  current_pillarId = atomicAdd(pillar_num, 1);
  if (current_pillarId > max_voxel_size - 1) return;

  voxel_num[current_pillarId] = count;

  uint3 idx = {0, voxel_idy, voxel_idx};
  ((uint3 *)voxel_idxs)[current_pillarId] = idx;

  for (int i = 0; i < count; i++) {
    int inIndex = voxel_index * MAX_POINT_IN_VOXEL_SIZE + i;
    int outIndex = current_pillarId * MAX_POINT_IN_VOXEL_SIZE + i;
    ((float4 *)voxel_features)[outIndex] = ((float4 *)voxels)[inIndex];
  }

  // clear buffer for next infer
  atomicExch(mask + voxel_index, 0);
}

// create 4 channels
hipError_t generateBaseFeatures_launch(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, int max_voxel_size,
  unsigned int * pillar_num, float * voxel_features, float * voxel_num, int * voxel_idxs,
  hipStream_t stream)
{
  dim3 threads = {32, 32};
  dim3 blocks = {
    (grid_x_size + threads.x - 1) / threads.x, (grid_y_size + threads.y - 1) / threads.y};

  generateBaseFeatures_kernel<<<blocks, threads, 0, stream>>>(
    mask, voxels, grid_y_size, grid_x_size, max_voxel_size, pillar_num, voxel_features, voxel_num,
    voxel_idxs);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateFeatures_kernel(
  const float * voxel_features, const float * voxel_num_points, const int * coords,
  const unsigned int * num_voxels, const float voxel_x, const float voxel_y, const float voxel_z,
  const float range_min_x, const float range_min_y, const float range_min_z, float * features)
{
  // voxel_features (float): (max_voxel_size, max_point_in_voxel_size, point_feature_size)
  // voxel_num_points (int): (max_voxel_size)
  // coords (int): (max_voxel_size, point_dim_size)
  int pillar_idx = blockIdx.x * WARPS_PER_BLOCK + threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;
  int point_idx = threadIdx.x % MAX_POINT_IN_VOXEL_SIZE;
  int pillar_idx_inBlock = threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;  // max_point_in_voxel_size

  unsigned int num_pillars = num_voxels[0];
  if (pillar_idx >= num_pillars) return;

  // load src
  __shared__ float4 pillarSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE];
  __shared__ float3 pillarSumSM[WARPS_PER_BLOCK];
  __shared__ int3 cordsSM[WARPS_PER_BLOCK];
  __shared__ int pointsNumSM[WARPS_PER_BLOCK];
  __shared__ float pillarOutSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE][ENCODER_IN_FEATURE_SIZE];

  if (threadIdx.x < WARPS_PER_BLOCK) {
    pointsNumSM[threadIdx.x] = voxel_num_points[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    cordsSM[threadIdx.x] = ((int3 *)coords)[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    pillarSumSM[threadIdx.x] = {0, 0, 0};
  }

  pillarSM[pillar_idx_inBlock][point_idx] =
    ((float4 *)voxel_features)[pillar_idx * MAX_POINT_IN_VOXEL_SIZE + point_idx];
  __syncthreads();

  // calculate sm in a pillar
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].x), pillarSM[pillar_idx_inBlock][point_idx].x);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].y), pillarSM[pillar_idx_inBlock][point_idx].y);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].z), pillarSM[pillar_idx_inBlock][point_idx].z);
  }
  __syncthreads();

  // feature-mean
  float3 mean;
  float validPoints = pointsNumSM[pillar_idx_inBlock];
  mean.x = pillarSumSM[pillar_idx_inBlock].x / validPoints;
  mean.y = pillarSumSM[pillar_idx_inBlock].y / validPoints;
  mean.z = pillarSumSM[pillar_idx_inBlock].z / validPoints;

  mean.x = pillarSM[pillar_idx_inBlock][point_idx].x - mean.x;
  mean.y = pillarSM[pillar_idx_inBlock][point_idx].y - mean.y;
  mean.z = pillarSM[pillar_idx_inBlock][point_idx].z - mean.z;

  // calculate offset
  float x_offset = voxel_x / 2 + cordsSM[pillar_idx_inBlock].z * voxel_x + range_min_x;
  float y_offset = voxel_y / 2 + cordsSM[pillar_idx_inBlock].y * voxel_y + range_min_y;
  float z_offset = voxel_z / 2 + cordsSM[pillar_idx_inBlock].x * voxel_z + range_min_z;

  // feature-offset
  float3 center;
  center.x = pillarSM[pillar_idx_inBlock][point_idx].x - x_offset;
  center.y = pillarSM[pillar_idx_inBlock][point_idx].y - y_offset;
  center.z = pillarSM[pillar_idx_inBlock][point_idx].z - z_offset;

  // store output
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    pillarOutSM[pillar_idx_inBlock][point_idx][0] = pillarSM[pillar_idx_inBlock][point_idx].x;
    pillarOutSM[pillar_idx_inBlock][point_idx][1] = pillarSM[pillar_idx_inBlock][point_idx].y;
    pillarOutSM[pillar_idx_inBlock][point_idx][2] = pillarSM[pillar_idx_inBlock][point_idx].z;
    pillarOutSM[pillar_idx_inBlock][point_idx][3] = pillarSM[pillar_idx_inBlock][point_idx].w;

    pillarOutSM[pillar_idx_inBlock][point_idx][4] = mean.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][5] = mean.y;
    pillarOutSM[pillar_idx_inBlock][point_idx][6] = mean.z;

    pillarOutSM[pillar_idx_inBlock][point_idx][7] = center.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][8] = center.y;

  } else {
    pillarOutSM[pillar_idx_inBlock][point_idx][0] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][1] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][2] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][3] = 0;

    pillarOutSM[pillar_idx_inBlock][point_idx][4] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][5] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][6] = 0;

    pillarOutSM[pillar_idx_inBlock][point_idx][7] = 0;
    pillarOutSM[pillar_idx_inBlock][point_idx][8] = 0;
  }

  __syncthreads();

  for (int i = 0; i < ENCODER_IN_FEATURE_SIZE; i++) {
    int outputSMId = pillar_idx_inBlock * MAX_POINT_IN_VOXEL_SIZE * ENCODER_IN_FEATURE_SIZE +
                     i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    int outputId = pillar_idx * MAX_POINT_IN_VOXEL_SIZE * ENCODER_IN_FEATURE_SIZE +
                   i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    features[outputId] = ((float *)pillarOutSM)[outputSMId];
  }
}

// cspell: ignore divup
hipError_t generateFeatures_launch(
  const float * voxel_features, const float * voxel_num_points, const int * coords,
  const unsigned int * num_voxels, const std::size_t max_voxel_size, const float voxel_size_x,
  const float voxel_size_y, const float voxel_size_z, const float range_min_x,
  const float range_min_y, const float range_min_z, float * features, hipStream_t stream)
{
  dim3 blocks(divup(max_voxel_size, WARPS_PER_BLOCK));
  dim3 threads(WARPS_PER_BLOCK * MAX_POINT_IN_VOXEL_SIZE);
  generateFeatures_kernel<<<blocks, threads, 0, stream>>>(
    voxel_features, voxel_num_points, coords, num_voxels, voxel_size_x, voxel_size_y, voxel_size_z,
    range_min_x, range_min_y, range_min_z, features);

  return hipGetLastError();
}

}  // namespace centerpoint
