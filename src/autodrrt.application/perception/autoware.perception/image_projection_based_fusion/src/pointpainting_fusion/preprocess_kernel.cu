#include "hip/hip_runtime.h"
// Copyright 2022 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "image_projection_based_fusion/pointpainting_fusion/preprocess_kernel.hpp"

#include <stdexcept>
// #include <lidar_centerpoint/utils.hpp>

namespace
{
const std::size_t MAX_POINT_IN_VOXEL_SIZE = 32;  // the same as max_point_in_voxel_size_ in config
const std::size_t WARPS_PER_BLOCK = 4;
const std::size_t ENCODER_IN_FEATURE_SIZE = 12;  // same as encoder_in_feature_size_ in config.hpp
const int POINT_FEATURE_SIZE = 7;

// cspell: ignore divup
std::size_t divup(const std::size_t a, const std::size_t b)
{
  if (a == 0) {
    throw std::runtime_error("A dividend of divup isn't positive.");
  }
  if (b == 0) {
    throw std::runtime_error("A divisor of divup isn't positive.");
  }

  return (a + b - 1) / b;
}

}  // namespace

namespace image_projection_based_fusion
{
__global__ void generateVoxels_random_kernel(
  const float * points, size_t points_size, float min_x_range, float max_x_range, float min_y_range,
  float max_y_range, float min_z_range, float max_z_range, float pillar_x_size, float pillar_y_size,
  float pillar_z_size, int grid_y_size, int grid_x_size, unsigned int * mask, float * voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_idx >= points_size) return;

  float x = points[point_idx * POINT_FEATURE_SIZE];
  float y = points[point_idx * POINT_FEATURE_SIZE + 1];
  float z = points[point_idx * POINT_FEATURE_SIZE + 2];

  if (
    x < min_x_range || x >= max_x_range || y < min_y_range || y >= max_y_range || z < min_z_range ||
    z >= max_z_range)
    return;

  int voxel_idx = floorf((x - min_x_range) / pillar_x_size);
  int voxel_idy = floorf((y - min_y_range) / pillar_y_size);
  unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if (point_id >= MAX_POINT_IN_VOXEL_SIZE) return;
  float * address =
    voxels + (voxel_index * MAX_POINT_IN_VOXEL_SIZE + point_id) * POINT_FEATURE_SIZE;
  for (unsigned int i = 0; i < POINT_FEATURE_SIZE; ++i) {
    atomicExch(address + i, points[point_idx * POINT_FEATURE_SIZE + i]);
  }
}

hipError_t generateVoxels_random_launch(
  const float * points, size_t points_size, float min_x_range, float max_x_range, float min_y_range,
  float max_y_range, float min_z_range, float max_z_range, float pillar_x_size, float pillar_y_size,
  float pillar_z_size, int grid_y_size, int grid_x_size, unsigned int * mask, float * voxels,
  hipStream_t stream)
{
  dim3 blocks((points_size + 256 - 1) / 256);
  dim3 threads(256);
  generateVoxels_random_kernel<<<blocks, threads, 0, stream>>>(
    points, points_size, min_x_range, max_x_range, min_y_range, max_y_range, min_z_range,
    max_z_range, pillar_x_size, pillar_y_size, pillar_z_size, grid_y_size, grid_x_size, mask,
    voxels);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateBaseFeatures_kernel(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, int max_voxel_size,
  unsigned int * pillar_num, float * voxel_features, float * voxel_num, int * voxel_idxs)
{
  unsigned int voxel_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int voxel_idy = blockIdx.y * blockDim.y + threadIdx.y;

  if (voxel_idx >= grid_x_size || voxel_idy >= grid_y_size) return;

  unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;
  unsigned int count = mask[voxel_index];
  if (!(count > 0)) return;
  count = count < MAX_POINT_IN_VOXEL_SIZE ? count : MAX_POINT_IN_VOXEL_SIZE;

  unsigned int current_pillarId = 0;
  current_pillarId = atomicAdd(pillar_num, 1);
  if (current_pillarId > max_voxel_size - 1) return;

  voxel_num[current_pillarId] = count;

  uint3 idx = {0, voxel_idy, voxel_idx};
  ((uint3 *)voxel_idxs)[current_pillarId] = idx;

  for (int i = 0; i < count; i++) {
    int inIndex = voxel_index * MAX_POINT_IN_VOXEL_SIZE + i;
    int outIndex = current_pillarId * MAX_POINT_IN_VOXEL_SIZE + i;
    for (unsigned int j = 0; j < POINT_FEATURE_SIZE; ++j) {
      voxel_features[outIndex * POINT_FEATURE_SIZE + j] = voxels[inIndex * POINT_FEATURE_SIZE + j];
    }
  }

  // clear buffer for next infer
  atomicExch(mask + voxel_index, 0);
}

// create 4 channels
hipError_t generateBaseFeatures_launch(
  unsigned int * mask, float * voxels, int grid_y_size, int grid_x_size, int max_voxel_size,
  unsigned int * pillar_num, float * voxel_features, float * voxel_num, int * voxel_idxs,
  hipStream_t stream)
{
  dim3 threads = {32, 32};
  dim3 blocks = {
    (grid_x_size + threads.x - 1) / threads.x, (grid_y_size + threads.y - 1) / threads.y};

  generateBaseFeatures_kernel<<<blocks, threads, 0, stream>>>(
    mask, voxels, grid_y_size, grid_x_size, max_voxel_size, pillar_num, voxel_features, voxel_num,
    voxel_idxs);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateFeatures_kernel(
  const float * voxel_features, const float * voxel_num_points, const int * coords,
  const unsigned int * num_voxels, const float voxel_x, const float voxel_y, const float voxel_z,
  const float range_min_x, const float range_min_y, const float range_min_z, float * features,
  const std::size_t encoder_in_feature_size)
{
  // voxel_features (float): (max_num_voxels, max_num_points_per_voxel, point_feature_size)
  // voxel_num_points (int): (max_num_voxels)
  // coords (int): (max_num_voxels, point_dim_size)
  int pillar_idx = blockIdx.x * WARPS_PER_BLOCK + threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;
  int point_idx = threadIdx.x % MAX_POINT_IN_VOXEL_SIZE;
  int pillar_idx_inBlock = threadIdx.x / MAX_POINT_IN_VOXEL_SIZE;

  unsigned int num_pillars = num_voxels[0];
  if (pillar_idx >= num_pillars) return;

  // load src
  __shared__ float pillarSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE][POINT_FEATURE_SIZE];
  __shared__ float3 pillarSumSM[WARPS_PER_BLOCK];
  __shared__ int3 cordsSM[WARPS_PER_BLOCK];
  __shared__ int pointsNumSM[WARPS_PER_BLOCK];
  __shared__ float pillarOutSM[WARPS_PER_BLOCK][MAX_POINT_IN_VOXEL_SIZE][ENCODER_IN_FEATURE_SIZE];

  if (threadIdx.x < WARPS_PER_BLOCK) {
    pointsNumSM[threadIdx.x] = voxel_num_points[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    cordsSM[threadIdx.x] = ((int3 *)coords)[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
    pillarSumSM[threadIdx.x] = {0, 0, 0};
  }

  for (std::size_t i = 0; i < POINT_FEATURE_SIZE; ++i) {
    pillarSM[pillar_idx_inBlock][point_idx][i] = voxel_features
      [(POINT_FEATURE_SIZE)*pillar_idx * MAX_POINT_IN_VOXEL_SIZE + (POINT_FEATURE_SIZE)*point_idx +
       i];
  }
  __syncthreads();

  // calculate sm in a pillar
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].x), pillarSM[pillar_idx_inBlock][point_idx][0]);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].y), pillarSM[pillar_idx_inBlock][point_idx][1]);
    atomicAdd(&(pillarSumSM[pillar_idx_inBlock].z), pillarSM[pillar_idx_inBlock][point_idx][2]);
  }
  __syncthreads();

  // feature-mean
  float3 mean;
  float validPoints = pointsNumSM[pillar_idx_inBlock];
  mean.x = pillarSumSM[pillar_idx_inBlock].x / validPoints;
  mean.y = pillarSumSM[pillar_idx_inBlock].y / validPoints;
  mean.z = pillarSumSM[pillar_idx_inBlock].z / validPoints;

  mean.x = pillarSM[pillar_idx_inBlock][point_idx][0] - mean.x;
  mean.y = pillarSM[pillar_idx_inBlock][point_idx][1] - mean.y;
  mean.z = pillarSM[pillar_idx_inBlock][point_idx][2] - mean.z;

  // calculate offset
  float x_offset = voxel_x / 2 + cordsSM[pillar_idx_inBlock].z * voxel_x + range_min_x;
  float y_offset = voxel_y / 2 + cordsSM[pillar_idx_inBlock].y * voxel_y + range_min_y;
  float z_offset = voxel_z / 2 + cordsSM[pillar_idx_inBlock].x * voxel_z + range_min_z;

  // feature-offset
  float3 center;
  center.x = pillarSM[pillar_idx_inBlock][point_idx][0] - x_offset;
  center.y = pillarSM[pillar_idx_inBlock][point_idx][1] - y_offset;
  center.z = pillarSM[pillar_idx_inBlock][point_idx][2] - z_offset;

  // store output
  if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
    for (std::size_t i = 0; i < POINT_FEATURE_SIZE; ++i) {
      pillarOutSM[pillar_idx_inBlock][point_idx][i] = pillarSM[pillar_idx_inBlock][point_idx][i];
    }

    // change index
    pillarOutSM[pillar_idx_inBlock][point_idx][POINT_FEATURE_SIZE] = mean.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][POINT_FEATURE_SIZE + 1] = mean.y;
    pillarOutSM[pillar_idx_inBlock][point_idx][POINT_FEATURE_SIZE + 2] = mean.z;

    pillarOutSM[pillar_idx_inBlock][point_idx][POINT_FEATURE_SIZE + 3] = center.x;
    pillarOutSM[pillar_idx_inBlock][point_idx][POINT_FEATURE_SIZE + 4] = center.y;

  } else {
    for (std::size_t i = 0; i < encoder_in_feature_size; ++i) {
      pillarOutSM[pillar_idx_inBlock][point_idx][i] = 0;
    }
  }

  __syncthreads();

  for (int i = 0; i < encoder_in_feature_size; i++) {
    int outputSMId = pillar_idx_inBlock * MAX_POINT_IN_VOXEL_SIZE * ENCODER_IN_FEATURE_SIZE +
                     i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    int outputId = pillar_idx * MAX_POINT_IN_VOXEL_SIZE * encoder_in_feature_size +
                   i * MAX_POINT_IN_VOXEL_SIZE + point_idx;
    features[outputId] = ((float *)pillarOutSM)[outputSMId];
  }
}

hipError_t generateFeatures_launch(
  const float * voxel_features, const float * voxel_num_points, const int * coords,
  const unsigned int * num_voxels, const std::size_t max_voxel_size, const float voxel_size_x,
  const float voxel_size_y, const float voxel_size_z, const float range_min_x,
  const float range_min_y, const float range_min_z, float * features,
  const std::size_t encoder_in_feature_size, hipStream_t stream)
{
  dim3 blocks(divup(max_voxel_size, WARPS_PER_BLOCK));
  dim3 threads(WARPS_PER_BLOCK * MAX_POINT_IN_VOXEL_SIZE);
  generateFeatures_kernel<<<blocks, threads, 0, stream>>>(
    voxel_features, voxel_num_points, coords, num_voxels, voxel_size_x, voxel_size_y, voxel_size_z,
    range_min_x, range_min_y, range_min_z, features, encoder_in_feature_size);

  return hipGetLastError();
}

}  // namespace image_projection_based_fusion
