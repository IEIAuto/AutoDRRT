#include "hip/hip_runtime.h"
// Copyright 2023 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// Copyright (c) OpenMMLab. All rights reserved.

#include "cuda_utils.hpp"
#include "grid_priors_kernel.hpp"
#include "trt_plugin_helper.hpp"

#include <hip/hip_fp16.h>

template <typename scalar_t>
__global__ void grid_priors_kernel(
  const scalar_t * base_anchor, scalar_t * output, int num_base_anchors, int feat_w, int feat_h,
  int stride_w, int stride_h)
{
  extern __shared__ scalar_t shared_base_anchor[];
  for (int i = threadIdx.x; i < num_base_anchors * 4; i += blockDim.x) {
    shared_base_anchor[i] = base_anchor[i];
  }
  __syncthreads();

  CUDA_1D_KERNEL_LOOP(index, num_base_anchors * feat_w * feat_h)
  {
    const int a_offset = (index % num_base_anchors) << 2;
    const scalar_t w = static_cast<scalar_t>(((index / num_base_anchors) % feat_w) * stride_w);
    const scalar_t h = static_cast<scalar_t>((index / (feat_w * num_base_anchors)) * stride_h);

    auto out_start = output + index * 4;
    out_start[0] = shared_base_anchor[a_offset] + w;
    out_start[1] = shared_base_anchor[a_offset + 1] + h;
    out_start[2] = shared_base_anchor[a_offset + 2] + w;
    out_start[3] = shared_base_anchor[a_offset + 3] + h;
  }
}

template <typename scalar_t>
void grid_priors_impl(
  const scalar_t * base_anchor, scalar_t * output, int num_base_anchors, int feat_w, int feat_h,
  int stride_w, int stride_h, hipStream_t stream)
{
  grid_priors_kernel<<<
    GET_BLOCKS(num_base_anchors * feat_w * feat_h), THREADS_PER_BLOCK,
    DIVUP(num_base_anchors * 4, 32) * 32 * sizeof(scalar_t), stream>>>(
    base_anchor, output, num_base_anchors, feat_w, feat_h, stride_w, stride_h);
}

template void grid_priors_impl<float>(
  const float * base_anchor, float * output, int num_base_anchors, int feat_w, int feat_h,
  int stride_w, int stride_h, hipStream_t stream);
